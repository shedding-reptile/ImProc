#include "hip/hip_runtime.h"

#include <cstring>
#include <iostream>

#define checkCudaErrors(err) __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char* file, const int line)
{
	if (hipSuccess != err)
	{
		std::cerr << file << "(" << line << ") : CUDA Runtime API error " << err << ": " << hipGetErrorString(err) << ".\n";
		exit(EXIT_FAILURE);
	}
}

__global__ void kernel(hipTextureObject_t tex, int width, int height, unsigned char* outputData)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		return;
	}

	outputData[y * width * 3 + 3 * x] = tex2D<unsigned char>(tex, 3 * x, y);
	outputData[y * width * 3 + 3 * x + 1] = tex2D<unsigned char>(tex, 3 * x + 1, y);
	outputData[y * width * 3 + 3 * x + 2] = tex2D<unsigned char>(tex, 3 * x + 2, y);
}

extern "C" void process(unsigned char* inBuffer, int width, int height, int channels, unsigned char** outBuffer, int &stride)
{
	size_t inputStride = sizeof(unsigned char) * width * channels;

	unsigned char* devImageIn = nullptr;
	size_t inPitch;
	hipError_t err = hipMallocPitch(&devImageIn, &inPitch, inputStride, height);
	err = hipMemcpy2D(devImageIn, inPitch, inBuffer, inputStride, inputStride, height, hipMemcpyHostToDevice);
	checkCudaErrors(err);

	unsigned char* devImageOut = nullptr;
	size_t outPitch;
	err = hipMallocPitch(&devImageOut, &outPitch, inputStride, height);
	checkCudaErrors(err);

	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypePitch2D;
	texRes.res.pitch2D.devPtr = devImageIn;
	texRes.res.pitch2D.desc = desc;
	texRes.res.pitch2D.width = static_cast<size_t>(width) * channels;
	texRes.res.pitch2D.height = height;
	texRes.res.pitch2D.pitchInBytes = inPitch;
	hipTextureDesc texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = false;
	texDescr.filterMode = hipFilterModePoint;
	texDescr.addressMode[0] = hipAddressModeWrap;
	texDescr.addressMode[1] = hipAddressModeWrap;
	texDescr.readMode = hipReadModeElementType;

	hipTextureObject_t texture;
	err = hipCreateTextureObject(&texture, &texRes, &texDescr, NULL);
	checkCudaErrors(err);

	dim3 blockSize(16, 16);
	dim3 gridSize(width + blockSize.x / blockSize.x, height + blockSize.y/ blockSize.y);
	
	kernel<<<gridSize, blockSize>>>(texture, width, height, devImageOut);
	stride = static_cast<int>(outPitch);
	*outBuffer = new unsigned char[stride * height];
	for (int i = 0; i < stride * height; i += 3)
	{
		(*outBuffer)[i] = 255;
		(*outBuffer)[i+1] = 0;
		(*outBuffer)[i+2] = 0;
	}
	err = hipMemcpy2D(*outBuffer, inPitch, devImageOut, inputStride, inputStride, height, hipMemcpyDeviceToHost);
	checkCudaErrors(err);

	hipDestroyTextureObject(texture);
	hipFree(devImageIn);
	hipFree(&inPitch);
	hipFree(&outPitch);
}
